#include "hip/hip_runtime.h"
#include <array>
#include <chrono>
#include <iostream>
#include <memory>
#include <numeric>
#include <random>
#include <system_error>
#include <type_traits>
#include <vector>

#include "err.cuh"
#include "matrix.cuh"
#include "transform.cuh"
#include "vector.cuh"

#include <sl/Core.hpp>

#include <opencv2/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/imgcodecs.hpp>

struct CudaDeleter {
    template <typename T>
    void operator()(T *ptr) noexcept {
        static_assert(std::is_trivially_destructible<T>::value, "T must be trivially destructible");

        hipFree(ptr);
    }
};

template <typename T>
class CudaAllocator {
public:
    static_assert(std::is_trivially_copyable<T>::value, "T must be trivially copyable");
    static_assert(std::is_trivially_destructible<T>::value, "T must be trivially destructible");

    using value_type = T;

    T* allocate(std::size_t n) {
        T *device_ptr;

        const std::error_code ec = hipMalloc(&device_ptr, n * sizeof(T));

        if (ec) {
            throw std::bad_alloc();
        }

        return device_ptr;
    }

    void deallocate(T *device_ptr, std::size_t) noexcept {
        hipFree(device_ptr);
    }

    template <typename ...Ts>
    void construct(T *device_ptr, Ts &&...ts) {
        const T host(std::forward<Ts>(ts)...);

        const std::error_code ec = hipMemcpy(
            device_ptr,
            std::addressof(host),
            sizeof(T),
            hipMemcpyHostToDevice
        );

        if (ec) {
            throw std::system_error(ec);
        }
    }

    void construct(T *device_ptr, const T &host) {
        const std::error_code ec = hipMemcpy(
            device_ptr,
            std::addressof(host),
            sizeof(T),
            hipMemcpyHostToDevice
        );

        if (ec) {
            throw std::system_error(ec);
        }
    }

    void construct(T *device_ptr) noexcept(std::is_trivially_default_constructible<T>::value) {
        do_construct_default(std::is_trivially_default_constructible<T>(), device_ptr);
    }

    void destroy(T*) noexcept {}

private:
    void do_construct_default(std::true_type, T*) noexcept {}

    void do_construct_default(std::false_type, T *device_ptr) {
        static const T host_default;

        const std::error_code ec = hipMemcpy(
            device_ptr,
            std::addressof(host_default),
            sizeof(T),
            hipMemcpyHostToDevice
        );

        if (ec) {
            throw std::system_error(ec);
        }
    }
};

template <typename T>
using CudaVector = std::vector<T, CudaAllocator<T>>;

template <typename T, std::enable_if_t<!std::is_array<T>::value, int> = 0>
std::unique_ptr<T, CudaDeleter> to_device(const T &host) {
    static_assert(std::is_trivially_copyable<T>::value, "");

    T *device_ptr;

    const std::error_code malloc_ec = hipMalloc(&device_ptr, sizeof(T));

    if (malloc_ec) {
        throw std::bad_alloc();
    }

    const std::error_code memcpy_ec = hipMemcpy(device_ptr, std::addressof(host), sizeof(T), hipMemcpyHostToDevice);

    if (memcpy_ec) {
        throw std::system_error(memcpy_ec);
    }

    return {device_ptr, CudaDeleter()};
}

template <typename T>
CudaVector<T> to_device(const T host[], std::size_t n) {
    static_assert(
        std::is_trivially_default_constructible<T>::value && std::is_trivially_copyable<T>::value,
        "T must be trivially default constructible and copyable"
    );

    CudaVector<T> device(n);

    const std::error_code ec = hipMemcpy(
        device.data(),
        host,
        n * sizeof(T),
        hipMemcpyHostToDevice
    );

    if (ec) {
        throw std::system_error(ec);
    }

    return device;
}

template <typename T, typename A>
CudaVector<T> to_device(const std::vector<T, A> &host) {
    return to_device(host.data(), host.size());
}

template <typename T, std::size_t N>
CudaVector<T> to_device(const std::array<T, N> &host) {
    return to_device(host.data(), N);
}

template <typename T, std::size_t N>
CudaVector<T> to_device(const T (&host)[N]) {
    return to_device(host, N);
}

template <typename T, typename A = std::allocator<T>>
std::vector<T, A> to_host(const std::vector<T, CudaAllocator<T>> &device) {
    std::vector<T, A> host(device.size());

    const std::error_code ec = hipMemcpy(
        host.data(),
        device.data(),
        device.size() * sizeof(T),
        hipMemcpyDeviceToHost
    );

    if (ec) {
        throw std::system_error(ec);
    }

    return host;
}

template <typename T>
T to_host(const std::unique_ptr<T, CudaDeleter> &ptr) {
    T host;

    const std::error_code ec = hipMemcpy(
        std::addressof(host),
        std::addressof(*ptr),
        sizeof(T),
        hipMemcpyDeviceToHost
    );

    if (ec) {
        throw std::system_error(ec);
    }

    return host;
}

constexpr std::size_t div_to_inf(std::size_t x, std::size_t y) noexcept {
    const std::size_t res = x / y;

    if (x % y != 0) {
        return res + 1;
    }

    return res;
}

__host__ __device__ sl::uchar4 from_packed(float x) {
    union Converter {
        float scalar;
        sl::uchar4 vector;
    };

    return Converter{x}.vector;
}

__host__ __device__ float pack(sl::uchar4 x) noexcept {
    union Converter {
        sl::uchar4 vector;
        float scalar;
    };

    return Converter{x}.scalar;
}

sl::Mat random_xyzrgba(std::size_t width, std::size_t height) {
    const std::size_t numel = width * height;
    sl::Mat m(width, height, sl::MAT_TYPE_32F_C4);

    const auto gen_ptr = std::make_unique<std::mt19937>();
    std::uniform_real_distribution<float> pos_dist(-10, 10);
    std::uniform_int_distribution<std::uint8_t> color_dist;

    const auto gen_pos = [&gen_ptr, &pos_dist] {
        return pos_dist(*gen_ptr);
    };

    const auto gen_col = [&gen_ptr, &color_dist] {
        return color_dist(*gen_ptr);
    };

    const auto arr = m.getPtr<sl::float4>();
    for (std::size_t i = 0; i < numel; ++i) {
        arr[i][0] = gen_pos();
        arr[i][1] = gen_pos();
        arr[i][2] = gen_pos();
        arr[i][3] = pack({255, 255, 255, 127});
    }

    m.updateGPUfromCPU();

    return m;
}

__host__ __device__ std::uint32_t pack_bgra(
    std::uint8_t b, std::uint8_t g,
    std::uint8_t r, std::uint8_t a
)  noexcept {
    union Converter {
        std::uint8_t arr[4];
        std::uint32_t scalar;
    };

    return Converter{{b, g, r, a}}.scalar;
}

/**
 *  @param input Points to an array of length n. Each element must be
 *               a 4-tuple (X, Y, Z, RGBA) corresponding to a depth
 *               map. The RGBA component is packed into a 32-bit float,
 *               with each element being an 8-bit unsigned integer.
 *  @param output Points to a matrix with m rows and p columns. Each
 *                element must be a 4-tuple (B, G, R, A). The elements
 *                should be stored contiguously in row-major order.
 *  @param resolution The resolution (in meters) of each pixel in
 *                    output, such that each pixel represents a
 *                    (resolution x resolution) square.
 *  @param x_offset The offset (in meters) between the center of the
 *                  matrix and its leftmost edge, such that the pixel
 *                  at (0, 0) is located in free space at (-x_offset,
 *                  -y_offset).
 *  @param y_offset The offset (in meters) between the center of the
 *                  matrix and its topmost edge, such that the pixel
 *                  at (0, 0) is located in free space at (-x_offset,
 *                  -y_offset).
 */
__global__ void transform(const Transform &tf, const sl::float4 *input, std::uint32_t n,
                          std::uint32_t *output, std::uint32_t m, std::uint32_t p,
                          float resolution, float x_offset, float y_offset) {
    const std::uint32_t pixel_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (pixel_idx >= n) {
        return;
    }

    const sl::float4 &elem = input[pixel_idx];
    const Vector transformed = tf({elem[0], elem[1], elem[2]});

    const float pixel_x = (transformed.x() + x_offset) / resolution;
    const float pixel_y = (transformed.y() + y_offset) / resolution;

    if (pixel_x < 0 || pixel_y < 0) {
        return;
    }

    const auto i = static_cast<std::uint32_t>(pixel_y); // row idx
    const auto j = static_cast<std::uint32_t>(pixel_x); // col idx

    if (i >= m || j >= p) {
        return;
    }

    const sl::uchar4 rgba = from_packed(elem[3]);
    const std::uint32_t output_idx = i * p + j;
    output[output_idx] = pack_bgra(rgba[2], rgba[1], rgba[0], rgba[3]);
}

int main(int argc, char* argv[]) {
    const Transform host_tf = {
        {
            0.5, -0.14644661, 0.85355339,
            0.5, 0.85355339, -0.14644661,
            0.70710678, 0.5, 0.5
        },
        {1, 0, 0}
    };

    const auto device_tf_ptr = to_device(host_tf);

    constexpr std::uint32_t WIDTH = 1280;
    constexpr std::uint32_t HEIGHT = 720;
    const sl::Mat input = random_xyzrgba(WIDTH, HEIGHT);

    constexpr float RESOLUTION = 0.05;
    constexpr float X_RANGE = 20;
    constexpr float Y_RANGE = 20;

    constexpr auto OUTPUT_ROWS = static_cast<std::uint32_t>(Y_RANGE / RESOLUTION);
    constexpr auto OUTPUT_COLS = static_cast<std::uint32_t>(X_RANGE / RESOLUTION);

    cv::cuda::GpuMat output(OUTPUT_COLS, OUTPUT_ROWS, CV_8UC4, cv::Scalar(0, 0, 0, 255));

    constexpr std::uint32_t NUMEL = WIDTH * HEIGHT;
    constexpr std::uint32_t BLOCKSIZE = 256;
    constexpr std::uint32_t NUM_BLOCKS = div_to_inf(NUMEL, BLOCKSIZE);

    hipDeviceSynchronize();
    const auto start = std::chrono::steady_clock::now();

    transform<<<NUM_BLOCKS, BLOCKSIZE>>>(
        *device_tf_ptr,
        input.getPtr<sl::float4>(sl::MEM_GPU),
        NUMEL,
        output.ptr<std::uint32_t>(),
        OUTPUT_ROWS,
        OUTPUT_COLS,
        RESOLUTION,
        X_RANGE / 2,
        Y_RANGE / 2
    );

    const cv::Mat output_host(output);

    hipDeviceSynchronize();
    const auto end = std::chrono::steady_clock::now();
    const std::chrono::duration<double> elapsed = end - start;

    std::cout << "elapsed: " << elapsed.count() << "s\n";

    cv::imwrite("output.png", output_host, {cv::IMWRITE_PNG_COMPRESSION, 9});
}
